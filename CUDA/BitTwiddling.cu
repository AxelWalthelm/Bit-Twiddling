#include "hip/hip_runtime.h"


#define PROFILE 0
#if PROFILE
#define IF_PROFILE(x) x
#else
#define IF_PROFILE(x)
#endif

#define CUDA_TIMER 0
#if CUDA_TIMER
#include "cuda_device_time.h"
#define IF_CUDA_TIMER(x) x
#else
#define IF_CUDA_TIMER(x)
#endif

#include "BitTwiddling.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "../parallel_for_items.hpp"

#if defined(NDEBUG) && !defined(__OPTIMIZE__)
#pragma message("warning: CUDA host code not optimized")
#endif


void semi_exhaustive_search_for_8bit_rev_cuda()
{
    int n_rep = 1; // 3
    int n_sel = 8; // 8
    int n_shi = 5; // 4-6

    generate_replicator replicate(n_rep);
    k_out_of_n_bits<uint32_t> select(n_sel, 32);
    k_out_of_n_bits<uint32_t> shift(n_shi, 32);
    const uint64_t N = generators::get_count(shift, select, replicate);
	printf("Total number of combinations: %" PRIu64 "\n", N);

    const int steps = 2000; // number of tests done in each thread; empiric value: threads run ca. 50 ms for 1000 steps; avoid timeout
    const int threads = 32; // threads per block; must be a multiple of 32; more than 32 is not better and even slightly bad
    const uint64_t blocks_total = div_ceil(N, threads * steps);
    const uint64_t blocks_x_max = 0x1000; //0x7fffffff; // CUDA 5 or higher: Maximum x-dimension of a grid of thread blocks [thread blocks] is 2^31-1
    const uint64_t iteration_blocks = std::min(blocks_total, blocks_x_max);
    const uint64_t iterations = div_ceil(blocks_total, iteration_blocks);
    printf("Total number of iterations: %" PRIu64 "\n", iterations);
    printf("Combinations per iteration: %" PRIu64 " >= %" PRIu64 "\n", iteration_blocks * threads * steps, div_ceil(N, iterations));
    parallel_for_range(378222, iterations,
        [=](uint64_t iteration)
        {
            auto start_time = GetHighResolutionTime();
            uint64_t start = iteration * iteration_blocks;
            semi_exhaustive_search_for_8bit_rev_cpu((uint32_t)iteration_blocks, threads, start, steps, n_rep, n_sel, n_shi);
            printf("%" PRIu64 "/%" PRIu64 " CPU in %.3fs\n", iteration, iterations, GetHighResolutionTimeElapsedNs(start_time) * 1e-9);
        },
        [=](uint64_t iteration)
        {
            auto start_time = GetHighResolutionTime();
            uint64_t start = iteration * iteration_blocks;
            CUDA_DO_SYNC(semi_exhaustive_search_for_8bit_rev_kernel<<<(uint32_t)iteration_blocks, threads>>>(start, steps, n_rep, n_sel, n_shi));
            printf("%" PRIu64 "/%" PRIu64 " GPU in %.3fs\n", iteration, iterations, GetHighResolutionTimeElapsedNs(start_time) * 1e-9);
        });

    printf("Done.");
}
